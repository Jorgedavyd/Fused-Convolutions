#include "hip/hip_runtime.h"
#include <torch/extensions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "fusedConv.cuh"

template <typename scalar_t>
__global__ void fwd1dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void fwd2dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void fwd3dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void bwd1dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
    };

template <typename scalar_t>
__global__ void bwd2dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void bwd3dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
static torch::Tensor Conv1D::forward (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        fwd1dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
static torch::autograd::variable_list Conv1D::backward (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        bwd1dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
static torch::Tensor Conv2D::forward (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        fwd2dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
static torch::autograd::variable_list Conv2D::backward (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        bwd2dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
static torch::Tensor Conv3D::forward (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        fwd3dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
static torch::autograd::variable_list Conv3D::backward (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        bwd3dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

