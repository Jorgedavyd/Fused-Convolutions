#include "hip/hip_runtime.h"
#include <torch/extensions.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "fusedConv.cuh"

template <typename scalar_t>
__global__ void fwd1dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void fwd2dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void fwd3dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void bwd1dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
    };

template <typename scalar_t>
__global__ void bwd2dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
__global__ void bwd3dKernel (
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> input,
    torch::PackedTensorAccessor32<scalar_t,3,torch::RestrictPtrTraits> weight,
) {
};

template <typename scalar_t>
std::vector<torch::Tensor> fwd_1D (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        fwd1dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
std::vector<torch::Tensor> bwd_1D (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        bwd1dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
std::vector<torch::Tensor> fwd_2D (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        fwd2dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
std::vector<torch::Tensor> bwd_2D (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        bwd2dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
std::vector<torch::Tensor> fwd_3D (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        fwd3dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

template <typename scalar_t>
std::vector<torch::Tensor> bwd_3D (
    torch::Tensor& input,
    torch::Tensor& weight,
    torch::Tensor& bias,
) {
        bwd3dKernel<scalar_t><<<BLOCKS, THREADS>>>
};

