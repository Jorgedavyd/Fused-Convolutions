#include <cstddef>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <functional>
#include "ptrTraits.cpp"

// Accessor for fused convolution
template <typename scalar_t, size_t N, template <typename U> class PtrTrait = RestrictedPtrTrait, typename index_t = uint32_t>
class Accessor  {
public:
    typedef typename PtrTrait<scalar_t>::PtrType PtrType;

    explicit __device__ Accessor (PtrType data_ptr, const index_t* stride, const index_t* size) : data_(data_ptr), stride_(stride), size_(size) {};

    __device__ Accessor (Accessor<scalar_t, N, PtrTrait, index_t>&& input) = delete;

    __device__ Accessor (Accessor<scalar_t, N, PtrTrait, index_t>& input) = delete;

    __device__ virtual const Accessor<scalar_t, N - 1, PtrTrait, index_t> operator[] (const index_t& idx) const {
        const index_t new_stride[N-1] = this->stride_ + 1;
        const index_t new_size[N-1] = this->size_ + 1;
        return Accessor<scalar_t,N - 1, PtrTrait, index_t>(this->data_ + this->stride_[0]*idx, new_stride, new_size);
    }

    __device__ virtual Accessor<scalar_t, N - 1, PtrTrait, index_t> operator[] (const index_t& idx) {
        index_t new_stride[N-1] = this->stride_ + 1;
        index_t new_size[N-1] = this->size_ + 1;
        return Accessor<scalar_t,N - 1, PtrTrait, index_t>(this->data_ + this->stride_[0]*idx, new_stride, new_size);
    }

    __device__ ~Accessor (void) = default;

private:
    PtrType data_;
    const index_t* stride_[N], size_[N];
};

template <typename scalar_t, template <typename U> class PtrTrait = RestrictedPtrTrait, typename index_t = uint32_t>
class Accessor<scalar_t, 1, PtrTrait, index_t> : public Accessor<scalar_t, 1, PtrTrait, index_t> {
public:
    typedef typename PtrTrait<scalar_t>::PtrType PtrType;

    explicit __device__ Accessor (PtrType data, index_t* stride, index_t* size) : data_(data), stride_(stride) {};

    // const-correctness
    __device__ virtual const PtrType operator[] (const index_t& idx) const {
        return this->data_ + this->stride[0]*idx;
    }
    __device__ virtual PtrType operator[] (const index_t& idx) {
        return this->data_ + this->stride[0]*idx;
    }

private:
    PtrType data_;
    const index_t stride[1];
};

template <typename scalar_t, size_t N, template <typename U> class PtrTrait = RestrictedPtrTrait, typename index_t = uint32_t>
class FFTAccessor : Accessor<scalar_t, N, PtrTrait, index_t> {
public:
    typedef typename PtrTrait<scalar_t>::PtrType PtrType;

    explicit __device__ FFTAccessor(PtrType data, index_t stride[N], index_t size[N]) : Accessor<scalar_t, N, PtrTrait, index_t> {};

};
